
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ker_butterfly_warp_reduce()
{
    int laneId = threadIdx.x & 0x1f;
    int value = 31 - laneId;

    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    printf("Thread %d final value = %d\n", threadIdx.x, value);
}

// why value of all threads is 32?
__global__ void ker_tree_warp_reduce()
{
    int tid = threadIdx.x;
    int val = 1;
    for(int i = 16; i > 0; i>>= 1)
    {
        val+=__shfl_down_sync(0xffffffff,val,i);
    }
    printf("Thread %d final value = %d\n", threadIdx.x, val);
}

__global__ void ker_test()
{
    int tid = threadIdx.x;
    int val = 1;
    if(tid < 16)
    {
        //why not dead lock
        val+=__shfl_down_sync(0xffffffff,val,1);
    }
    if(tid<16)
    {
        val+=1;
        // why not dead lock
        __syncwarp();
    }
    printf("Thread %d final value = %d\n", threadIdx.x, val);
}

int main() {
    // ker_butterfly_warp_reduce<<< 1, 32 >>>();
    // ker_tree_warp_reduce<<< 1, 32 >>>();

    ker_test<<<1,64>>>();
    hipDeviceSynchronize();

    return 0;
}
