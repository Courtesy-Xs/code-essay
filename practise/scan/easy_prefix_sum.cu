
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(res) if(res!=hipSuccess) { printf("Error: %s%d\n",__FILE__,__LINE__);   \
printf("Error code: %d, Reason: %s\n",res,hipGetErrorString(res));}

const int kGridSize = 1;
const int kBlockSize = 1024;

__global__ void ker_scan(int* src, int len)
{
    __shared__ int shm[1024];

    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    shm[tid] = src[tid];

    __syncthreads();

    int tmp = 0;
    for(int i = 1;i<len;i<<=1)
    {
        tmp = tid+i<len? shm[tid+i]:0;
        __syncthreads();
        shm[tid]+=tmp;
        __syncthreads();
    }

    src[tid] = shm[tid];

}


void destory(int* harr, int* darr)
{
    delete harr;
    hipFree(darr);
}


int main()
{
    int *harr = nullptr;
    int *darr = nullptr;
    const int numel = 1024;

    harr = (int*)malloc(numel*sizeof(int));
    CUDA_CHECK(hipMalloc((void**)(&darr),numel*sizeof(int)));

    for(int i = 0; i < numel; ++i)
    {
        *(harr+i) = 1;
    }

    CUDA_CHECK(hipMemcpy(darr,harr,sizeof(int)*numel,hipMemcpyHostToDevice));

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    ker_scan<<<kGridSize,kBlockSize,0,0>>>(darr,numel);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time,start,stop);
    float bandwidth = float(numel*sizeof(int))/1024/1024/(elapsed_time)*1000;

    hipMemcpy(harr,darr,sizeof(int)*1024,hipMemcpyDeviceToHost);
    for(int i = 0;i < numel;++i)
    {
        printf("harr[%d] = %d\t",i,harr[i]);
    }

    printf("cost time is %fms\n",elapsed_time);
    printf("bandWidth is %fGB/s\n",bandwidth);

    destory(harr,darr);
    return 1;
}
