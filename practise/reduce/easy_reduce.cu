
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(res) if(res!=hipSuccess) { printf("Error: %s%d\n",__FILE__,__LINE__);   \
printf("Error code: %d, Reason: %s\n",res,hipGetErrorString(res));}

const int kGridSize = 1;
const int kBlockSize = 1024;

__global__ void ker_reduce(int* src, int len)
{
    __shared__ int shm[1024];

    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    shm[tid] = src[tid];

    __syncthreads();

    for(int i = len>>1; i > 0 ; i>>=1)
    {
        if(tid < i)
        {
            shm[tid] += shm[tid+i];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        src[0] = shm[0];
    }
}


void destory(int* harr, int* darr)
{
    delete harr;
    hipFree(darr);
}

int main()
{
    int *harr = nullptr;
    int *darr = nullptr;
    const int numel = 1024;

    harr = (int*)malloc(numel*sizeof(int));
    CUDA_CHECK(hipMalloc((void**)(&darr),numel*sizeof(int)));

    for(int i = 0; i < numel; ++i)
    {
        *(harr+i) = 1;
    }

    CUDA_CHECK(hipMemcpy(darr,harr,sizeof(int)*numel,hipMemcpyHostToDevice));

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    ker_reduce<<<kGridSize,kBlockSize,0,0>>>(darr,numel);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time,start,stop);
    float bandwidth = float(numel*sizeof(int))/1024/1024/(elapsed_time)*1000;

    hipMemcpy(harr,darr,sizeof(int)*1024,hipMemcpyDeviceToHost);
    printf("sum is %d\n",harr[0]);
    printf("cost time is %fms\n",elapsed_time);
    printf("bandWidth is %fGB/s\n",bandwidth);

    destory(harr,darr);
    return 1;
}
