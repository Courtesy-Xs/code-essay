#include <hip/hip_runtime.h>
#include <stdio.h>

const int kBlockSize = 256;
const int kNumWarp = kBlockSize/32;
const int KGridSize = 4;

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}

__device__ int warp_reduce(int x)
{
    #pragma unroll
    for(int i = 16; i>0; i>>=1)
    {
        x+=__shfl_down_sync(0xffffffff,x,i);
    }
    return x;
}

__device__ int block_reduce(int* shm)
{
    int tid = threadIdx.x;
    int warp_id = tid/32;
    int lane_id = tid&0x1f;

    int val = shm[tid];
    int sum = 0;
    val=warp_reduce(val);

    __syncthreads();
    if(lane_id == 0)
    {
        shm[warp_id] = val;
    }
    if(warp_id==0)
    {
        sum = shm[warp_id];
        sum += __shfl_down_sync(0xf,sum,4);
        sum += __shfl_down_sync(0xf,sum,2);
        sum += __shfl_down_sync(0xf,sum,1);
    }

    return sum;
}

__device__ unsigned int count = 0;
__global__ void ker_reduce(int* src, int len, volatile int* result, unsigned int* k_count)
{
    __shared__ int shm[256];

    int tid = threadIdx.x;
    __shared__ unsigned is_last_block;

    shm[tid] = src[blockIdx.x*blockDim.x+tid];

    __syncthreads();

    result[blockIdx.x] = block_reduce(shm);

    __threadfence();

    if(threadIdx.x == 0)
    {
        is_last_block = atomicInc(&count,4);
    }

    __syncwarp();
    // __syncthreads();
    // 这个是用来sync is_last_block的，但是我这里最后只要第一个warp计算，所以没必要了，warp有隐式同步了
    // 考虑到indepentent thread schedule 的问题，还是加个sync_warp

    if(is_last_block == 3)
    {
        if(tid<8)
        {
            int val = result[tid];
            val+=__shfl_down_sync(0xf,val,2,2);
            val+=__shfl_down_sync(0x3,val,1,1);
            if(tid == 0)
            {
                src[0] = val;
            }
        }

    }
}

void initialize(int** harr, int** darr, int size)
{
    (*harr) = (int*)malloc(size*sizeof(int));
    CHECK(hipMalloc((void**)(darr),size*sizeof(int)));

    for(int i = 0; i < size; ++i)
    {
        (*harr[i]) = 1;
    }
    CHECK(hipMemcpy(darr,harr,sizeof(int)*size,hipMemcpyHostToDevice));
}

void destory(int* harr, int* darr)
{
    delete harr;
    hipFree(darr);
}

int main()
{
    int *harr = nullptr;
    int *darr = nullptr;
    int *dres = nullptr;
    const int numel = 1024;
    unsigned int count = 0;

    harr = (int*)malloc(numel*sizeof(int));
    CHECK(hipMalloc((void**)(&darr),numel*sizeof(int)));
    CHECK(hipMalloc((void**)(&dres),4*sizeof(int)));
    CHECK(hipMemset(dres,0,4*sizeof(int)));


    for(int i = 0; i < numel; ++i)
    {
        *(harr+i) = 1;
    }

    CHECK(hipMemcpy(darr,harr,sizeof(int)*numel,hipMemcpyHostToDevice));

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    ker_reduce<<<KGridSize,kBlockSize,0,0>>>(darr,numel,dres,&count);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time,start,stop);
    float bandwidth = float(numel*sizeof(int))/1024/1024/(elapsed_time)*1000;

    hipMemcpy(harr,darr,sizeof(int)*1024,hipMemcpyDeviceToHost);
    printf("sum is %d\n",harr[0]);
    printf("cost time is %fms\n",elapsed_time);
    printf("bandWidth is %fGB/s\n",bandwidth);

    destory(harr,darr);
    return 1;
}
