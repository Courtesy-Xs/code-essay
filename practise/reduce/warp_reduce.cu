
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ker_butterfly_warp_reduce()
{
    int laneId = threadIdx.x & 0x1f;
    // Seed starting value as inverse lane ID
    int value = 31 - laneId;

    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    printf("Thread %d final value = %d\n", threadIdx.x, value);
}

__global__ void ker_tree_warp_reduce()
{
    int tid = threadIdx.x;
    int val = 1;
    for(int i = 16; i > 0; i>>= 1)
    {
        val+=__shfl_down_sync(0xffffffff,val,i);
    }
    printf("Thread %d final value = %d\n", threadIdx.x, val);
}

__global__ void ker_test(int* data)
{
    int tid = threadIdx.x;
    int val = 1;
    if(tid < 16)
    {
        val+=__shfl_down_sync(0xffffffff,val,1);
    }
    printf("Thread %d final value = %d\n", threadIdx.x, val);
}

int main() {
    // ker_butterfly_warp_reduce<<< 1, 32 >>>();
    ker_tree_warp_reduce<<< 1, 32 >>>();

    // ker_test<<<1,64>>>();
    hipDeviceSynchronize();

    return 0;
}
