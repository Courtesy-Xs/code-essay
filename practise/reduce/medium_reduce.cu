#include <hip/hip_runtime.h>
#include <stdio.h>


const int kBlockSize = 256;
const int KGridSize = 1;

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__device__ int warp_reduce(int x)
{
    #pragma unroll
    for(int i = 16; i>0; i>>=1)
    {
        x+=__shfl_down_sync(0xffffffff,x,i);
    }
    return x;
}


__global__ void ker_reduce(int* src, int len)
{
    __shared__ int shm[256];
    __shared__ int warp_shm[8];

    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int warp_id = tid/32;
    int lane_id = tid%32;
    for(int i = tid; i < len; i+=2*(blockDim.x*gridDim.x))
    {
        shm[tid] += src[i] + src[i+blockDim.x*gridDim.x];
    }

    __syncthreads();

    int val = shm[tid];
    val = warp_reduce(val);
    if(lane_id == 0)
    {
        warp_shm[warp_id] = val;
    }

    __syncthreads();

    if(tid<8)
    {
        int val = warp_shm[tid];
        val += __shfl_down_sync(0xff,val,4);
        val += __shfl_down_sync(0xf,val,2);
        val += __shfl_down_sync(0x3,val,1);
        if(tid==0)
        {
            src[0] = val;
        }
    }
}

void destory(int* harr, int* darr)
{
    delete harr;
    hipFree(darr);
}

int main()
{
    int *harr = nullptr;
    int *darr = nullptr;
    const int numel = 1024;

    harr = (int*)malloc(numel*sizeof(int));
    CHECK(hipMalloc((void**)(&darr),numel*sizeof(int)));

    for(int i = 0; i < numel; ++i)
    {
        *(harr+i) = 1;
    }

    CHECK(hipMemcpy(darr,harr,sizeof(int)*numel,hipMemcpyHostToDevice));

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    ker_reduce<<<KGridSize,kBlockSize,0,0>>>(darr,numel);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time,start,stop);
    float bandwidth = float(numel*sizeof(int))/1024/1024/(elapsed_time)*1000;

    hipMemcpy(harr,darr,sizeof(int)*1024,hipMemcpyDeviceToHost);
    printf("sum is %d\n",harr[0]);
    printf("cost time is %fms\n",elapsed_time);
    printf("bandWidth is %fGB/s\n",bandwidth);

    destory(harr,darr);
    return 1;
}
